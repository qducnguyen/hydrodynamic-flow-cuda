#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

#define nx 					32
#define ny 					32
#define nt 					10000
#define nit 				50
#define c 					1.0
#define xmax 				2.0
#define ymax 				2.0
#define rho 				1.0
#define nu 					0.1
#define dt 					0.001
#define result_file_name 	"flow_results_cuda.txt"
#define display_num 		10
#define is_log 				0
#define log_file_name 		"flow_logs_cuda.log"
#define log_step 			50 


const int display_step = nt / display_num;


#define BlockSizeX 8 
#define BlockSizeY 8
#define GridSizeX nx / BlockSizeX
#define GridSizeY ny / BlockSizeY


void save_log(double *u, double *v, double *p, FILE *file, double dx, double dy, int step);
void save_results(double *u, double *v, double *p, const char *filename, double dx, double dy);


void init(double *u, double *v, double *p, double *pn, double *b)
{
	printf("%d", GridSizeX);
	int i, j;
	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			*(u + i * nx + j) = 0;
			*(v + i * nx + j) = 0;
			*(p + i * nx + j) = 0;
			*(pn + i * nx + j) = 0;
			*(b + i * nx + j) = 0;
		}
	}
}

__global__ void build_up_b(double *b, double *u, double *v, double dx, double dy)
{

	__shared__ double tile_u[BlockSizeY + 2][BlockSizeX + 2];
	__shared__ double tile_v[BlockSizeY + 2][BlockSizeX + 2];

	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < ny && j < nx)
	{
		tile_u[ty + 1][tx + 1] = *(u + i * nx + j);
		tile_v[ty + 1][tx + 1] = *(v + i * nx + j);
	}
	if (ty == 0 && i > 0)
	{
		tile_u[ty][tx + 1] = *(u + (i - 1) * nx + j);
		tile_v[ty][tx + 1] = *(v + (i - 1) * nx + j);
	}
	if (ty == BlockSizeY - 1 && i < ny - 1)
	{
		tile_u[ty + 2][tx + 1] = *(u + (i + 1) * nx + j);
		tile_v[ty + 2][tx + 1] = *(v + (i + 1) * nx + j);
	}
	if (tx == 0 && j > 0)
	{
		tile_u[ty + 1][tx] = *(u + i * nx + j - 1);
		tile_v[ty + 1][tx] = *(v + i * nx + j - 1);
	}
	if (tx == BlockSizeX - 1 && j < nx - 1)
	{
		tile_u[ty + 1][tx + 2] = *(u + i * nx + j + 1);
		tile_v[ty + 1][tx + 2] = *(v + i * nx + j + 1);
	}

	__syncthreads();

	if (i > 0 && i < ny - 1 && j > 0 && j < nx - 1)
	{
		*(b + i * nx + j) = rho * (1 / dt *
									   ((tile_u[ty + 1][tx + 2] - tile_u[ty + 1][tx]) / (2 * dx) + (tile_v[ty + 2][tx + 1] - tile_v[ty][tx + 1]) / (2 * dy)) -
								   (tile_u[ty + 1][tx + 2] - tile_u[ty + 1][tx]) * (tile_u[ty + 1][tx + 2] - tile_u[ty + 1][tx]) / (2 * 2 * dx * dx) -
								   2 * ((tile_u[ty + 2][tx + 1] - tile_u[ty][tx + 1]) / (2 * dy) *
										(tile_v[ty + 1][tx + 2] - tile_v[ty + 1][tx]) / (2 * dx)) -
								   (tile_v[ty + 2][tx + 1] - tile_v[ty][tx + 1]) * (tile_v[ty + 2][tx + 1] - tile_v[ty][tx + 1]) / (2 * 2 * dy * dy));
	}
}

__global__ void solve_pressure_poisson(double *p, double *pn, double *b, double dx, double dy)
{

	__shared__ double tile[BlockSizeY + 2][BlockSizeX + 2];

	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < ny && j < nx)
	{
		tile[ty + 1][tx + 1] = *(pn + i * nx + j);
	}
	if (ty == 0 && i > 0)
	{
		tile[ty][tx + 1] = *(pn + (i - 1) * nx + j);
	}
	if (ty == BlockSizeY - 1 && i < ny - 1)
	{
		tile[ty + 2][tx + 1] = *(pn + (i + 1) * nx + j);
	}
	if (tx == 0 && j > 0)
	{
		tile[ty + 1][tx] = *(pn + i * nx + j - 1);
	}
	if (tx == BlockSizeX - 1 && j < nx - 1)
	{
		tile[ty + 1][tx + 2] = *(pn + i * nx + j + 1);
	}

	__syncthreads();

	if (i > 0 && i < ny - 1 && j > 0 && j < nx - 1)
	{
		*(p + i * nx + j) = ((tile[ty + 1][tx + 2] + tile[ty + 1][tx]) * dy * dy +
							 (tile[ty + 2][tx + 1] + tile[ty][tx + 1]) * dx * dx) /
								(2 * (dx * dx + dy * dy)) -
							dx * dx * dy * dy / (2 * (dx * dx + dy * dy)) * *(b + i * nx + j);
	}

	if (j == nx - 1)
	{
		*(p + i * nx + j) = *(p + i * nx + j - 1);
	}

	if (i == 0)
	{
		*(p + i * nx + j) = *(p + (i + 1) * nx + j);
	}

	if (j == 0)
	{
		*(p + i * nx + j) = *(p + i * nx + j + 1);
	}

	if (i == ny - 1)
	{
		*(p + i * nx + j) = 0;
	}
}

__global__ void velocity_update(double *u, double *v, double *un, double *vn, double *p, double dx, double dy)
{

	__shared__ double tile_u[BlockSizeY + 2][BlockSizeX + 2];
	__shared__ double tile_v[BlockSizeY + 2][BlockSizeX + 2];

	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < ny && j < nx)
	{
		tile_u[ty + 1][tx + 1] = *(un + i * nx + j);
		tile_v[ty + 1][tx + 1] = *(vn + i * nx + j);
	}
	if (ty == 0 && i > 0)
	{
		tile_u[ty][tx + 1] = *(un + (i - 1) * nx + j);
		tile_v[ty][tx + 1] = *(vn + (i - 1) * nx + j);
	}
	if (ty == BlockSizeY - 1 && i < ny - 1)
	{
		tile_u[ty + 2][tx + 1] = *(un + (i + 1) * nx + j);
		tile_v[ty + 2][tx + 1] = *(vn + (i + 1) * nx + j);
	}
	if (tx == 0 && j > 0)
	{
		tile_u[ty + 1][tx] = *(un + i * nx + j - 1);
		tile_v[ty + 1][tx] = *(vn + i * nx + j - 1);
	}
	if (tx == BlockSizeX - 1 && j < nx - 1)
	{
		tile_u[ty + 1][tx + 2] = *(un + i * nx + j + 1);
		tile_v[ty + 1][tx + 2] = *(vn + i * nx + j + 1);
	}

	__syncthreads();

	if (i > 0 && i < ny - 1 && j > 0 && j < nx - 1)
	{

		*(u + i * nx + j) = tile_u[ty + 1][tx + 1] - tile_u[ty + 1][tx + 1] * dt / dx * (tile_u[ty + 1][tx + 1] - tile_u[ty + 1][tx]) -
							tile_v[ty + 1][tx + 1] * dt / dy *
								(tile_u[ty + 1][tx + 1] - tile_u[ty][tx + 1]) -
							dt / (2 * rho * dx) * (*(p + i * nx + j + 1) - *(p + i * nx + j - 1)) +
							nu * (dt / (dx * dx) *
									  (tile_u[ty + 1][tx + 2] - 2 * tile_u[ty + 1][tx + 1] + tile_u[ty + 1][tx]) +
								  dt / (dy * dy) *
									  (tile_u[ty + 2][tx + 1] - 2 * tile_u[ty + 1][tx + 1] + tile_u[ty][tx + 1]));

		*(v + i * nx + j) = tile_v[ty + 1][tx + 1] - tile_u[ty + 1][tx + 1] * dt / dx * (tile_v[ty + 1][tx + 1] - tile_v[ty + 1][tx]) -
							tile_v[ty + 1][tx + 1] * dt / dy *
								(tile_v[ty + 1][tx + 1] - tile_v[ty][tx + 1]) -
							dt / (2 * rho * dy) * (*(p + (i + 1) * nx + j) - *(p + (i - 1) * nx + j)) +
							nu * (dt / (dx * dx) *
									  (tile_v[ty + 1][tx + 2] - 2 * tile_v[ty + 1][tx + 1] + tile_v[ty + 1][tx]) +
								  dt / (dy * dy) *
									  (tile_v[ty + 2][tx + 1] - 2 * tile_v[ty + 1][tx + 1] + tile_v[ty][tx + 1]));
	}


	if (j == 0)
	{
		*(u + i * nx + j) = 0;
		*(v + i * nx + j) = 0;
	}

	if (j == nx - 1)
	{
		*(u + i * nx + j) = 0;
		*(v + i * nx + j) = 0;
	}

	if (i == 0)
	{
		*(u + i * nx + j) = 0;
		*(v + i * nx + j) = 0;
	}

	if (i == ny - 1)
	{
		*(u + i * nx + j) = 1; // set velocity on cavity lid equal to 1
		*(v + i * nx + j) = 0;
	}
}

int main()
{

	int n, it;

	double *ucpu, *vcpu, *pcpu, *bcpu, *pncpu;
	double dx = xmax / (nx - 1);
	double dy = ymax / (ny - 1);

	struct timeval time_start;
	struct timeval time_end;

	ucpu = (double *)malloc((nx * ny) * sizeof(double));
	vcpu = (double *)malloc((nx * ny) * sizeof(double));
	pcpu = (double *)malloc((nx * ny) * sizeof(double));
	pncpu = (double *)malloc((nx * ny) * sizeof(double));
	bcpu = (double *)malloc((nx * ny) * sizeof(double));

	init(ucpu, vcpu, pcpu, pncpu, bcpu);

	gettimeofday(&time_start, NULL);

	double *ugpu, *vgpu, *ungpu, *vngpu, *pgpu, *bgpu, *pngpu, *tempgpu, *tempgpu2;

	hipMalloc((void **)&ugpu, (nx * ny) * sizeof(double));
	hipMalloc((void **)&vgpu, (nx * ny) * sizeof(double));
	hipMalloc((void **)&ungpu, (nx * ny) * sizeof(double));
	hipMalloc((void **)&vngpu, (nx * ny) * sizeof(double));

	hipMalloc((void **)&pgpu, (nx * ny) * sizeof(double));
	hipMalloc((void **)&pngpu, (nx * ny) * sizeof(double));
	hipMalloc((void **)&tempgpu, (nx * ny) * sizeof(double));
	hipMalloc((void **)&tempgpu2, (nx * ny) * sizeof(double));

	hipMalloc((void **)&bgpu, (nx * ny) * sizeof(double));


	hipMemcpy(ugpu, ucpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vgpu, vcpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(ungpu, ucpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vngpu, vcpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(pgpu, pcpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(pngpu, pncpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(bgpu, bcpu, (nx * ny) * sizeof(double), hipMemcpyHostToDevice);

	dim3 dimGrid(GridSizeX, GridSizeY);
	dim3 dimBlock(BlockSizeX, BlockSizeY);

	for (n = 0; n < nt; n++)
	{
		build_up_b<<<dimGrid, dimBlock>>>(bgpu, ugpu, vgpu, dx, dy);

		for (it = 0; it < nit; it++)
		{
			tempgpu = pngpu;
			pngpu = pgpu;
			pgpu = tempgpu;
			solve_pressure_poisson<<<dimGrid, dimBlock>>>(pgpu, pngpu, bgpu, dx, dy);
		}


		tempgpu  = vngpu;
		tempgpu2 = ungpu;
		vngpu = vgpu;
		ungpu = ugpu;
		vgpu = tempgpu;
		ugpu = tempgpu2;

		velocity_update<<<dimGrid, dimBlock>>>(ugpu, vgpu, ungpu, vngpu, pgpu, dx, dy);

		if (n != 0 && ((n+1) % display_step) == 0){
			fprintf(stdout, "Running: %d / %d ... \n", n+1, nt);
		}


		if (n == nt - 1 && (nt % display_step) != 0){
			fprintf(stdout, "Running: %d / %d ... \n", n+1, nt);
		}

		if (is_log){
			FILE *filelog;
			hipMemcpy(ucpu, ugpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(vcpu, vgpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(pcpu, pgpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);

			if (n == 0){
				filelog = fopen(log_file_name, "w");
				save_log(ucpu, vcpu, pcpu, filelog, dx, dy, 1);
			}
			else if ( (n+1) % log_step == 0)
					save_log(ucpu, vcpu, pcpu, filelog, dx, dy, n+1);
			else if ( n == nt -1){
				if ((n + 1) % log_step != 0){
					save_log(ucpu, vcpu, pcpu, filelog, dx, dy, n+1);
				}
				fclose(filelog);
			}
		}
	}

	hipMemcpy(ucpu, ugpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(vcpu, vgpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(pcpu, pgpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(bcpu, bgpu, (nx * ny) * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(ugpu);
	hipFree(vgpu);
	hipFree(pgpu);
	hipFree(bgpu);
	hipFree(tempgpu);
	hipFree(tempgpu2);
	hipFree(pngpu);
	hipFree(ungpu);
	hipFree(vngpu);

	gettimeofday(&time_end, NULL);

	save_results(ucpu, vcpu, pcpu, result_file_name, dx, dy);

	free(ucpu);
	free(vcpu);
	free(pcpu);
	free(bcpu);

	double exec_time = (double)(time_end.tv_sec - time_start.tv_sec) +
					   (double)(time_end.tv_usec - time_start.tv_usec) / 1000000.0;

	printf("Running time for CUDA code: %lf\n", exec_time);

	return 0;
}

void save_log(double *u, double *v, double *p, FILE *file, double dx, double dy, int step)
{
	int i, j;

	if (step == 1){

		fprintf(file, "%d\n", nx);
		fprintf(file, "%d\n", ny);
		fprintf(file, "%d\n", nt);
		fprintf(file, "%d\n", nit);
		fprintf(file, "%f\n", c);
		fprintf(file, "%f\n", xmax);
		fprintf(file, "%f\n", ymax);
		fprintf(file, "%f\n", rho);
		fprintf(file, "%f\n", nu);
		fprintf(file, "%f\n", dt);
		fprintf(file, "%f\n", dx);
		fprintf(file, "%f\n", dy);
	}

	fprintf(file, "%d\n", step);

	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			// 20 pecision ..
			fprintf(file, "%.20lf ", *(u + i * nx + j));
		}
		fprintf(file, "\n");
	}

	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			// 20 pecision ..
			fprintf(file, "%.20lf ", *(v + i * nx + j));
		}
		fprintf(file, "\n");
	}

	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			// 20 pecision ..
			fprintf(file, "%.20lf ", *(p + i * nx + j));
		}
		fprintf(file, "\n");
	}


}


void save_results(double *u, double *v, double *p, const char *filename, double dx, double dy)
{
	//
	FILE *file = fopen(filename, "w");
	int i, j;

	fprintf(file, "%d\n", nx);
	fprintf(file, "%d\n", ny);
	fprintf(file, "%d\n", nt);
	fprintf(file, "%d\n", nit);
	fprintf(file, "%f\n", c);
	fprintf(file, "%f\n", xmax);
	fprintf(file, "%f\n", ymax);
	fprintf(file, "%f\n", rho);
	fprintf(file, "%f\n", nu);
	fprintf(file, "%f\n", dt);
	fprintf(file, "%f\n", dx);
	fprintf(file, "%f\n", dy);

	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			// 73 pecision ..
			fprintf(file, "%.73lf ", *(u + i * nx + j));
		}
		fprintf(file, "\n");
	}

	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			// 73 pecision ..
			fprintf(file, "%.73lf ", *(v + i * nx + j));
		}
		fprintf(file, "\n");
	}

	for (i = 0; i < ny; i++)
	{
		for (j = 0; j < nx; j++)
		{
			// 73 pecision ..
			fprintf(file, "%.73lf ", *(p + i * nx + j));
		}
		fprintf(file, "\n");
	}

	fclose(file);
}
