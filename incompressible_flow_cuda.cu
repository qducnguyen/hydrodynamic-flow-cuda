#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>


#define nx 32
#define ny 32
#define nt 10000
#define nit 50 
#define c 1.0
#define xmax 2.0
#define ymax 2.0 
#define rho 1.0 
#define nu 0.1
#define dt 0.001

#define GridSizeX 2
#define GridSizeY 2
#define BlockSizeX nx / GridSizeX
#define BlockSizeY ny / GridSizeY

void save_results(double *u, double *v, double *p, char *filename, double dx, double dy){
	//  
	FILE *file = fopen(filename, "w");
	int i,j;

	fprintf(file, "%d\n", nx);
	fprintf(file, "%d\n", ny);
	fprintf(file, "%d\n", nt);
	fprintf(file, "%d\n", nit);
	fprintf(file, "%f\n", c);
	fprintf(file, "%f\n", xmax);
	fprintf(file, "%f\n", ymax);
	fprintf(file, "%f\n", rho);
	fprintf(file, "%f\n", nu);
	fprintf(file, "%f\n", dt);
	fprintf(file, "%f\n", dx);
	fprintf(file, "%f\n", dy);


	// for (i = 0; i < ny; i++){
	// 	for (j = 0; j < nx; j++){
	// 		// 73 pecision ..
	// 		fprintf(file, "%.73lf ", *(u + i*nx + j));
	// 	}
	// 	fprintf(file, "\n");
	// }

	// for (i = 0; i < ny; i++){
	// 	for (j = 0; j < nx; j++){
	// 		// 73 pecision ..
	// 		fprintf(file, "%.73lf ", *(v + i*nx + j));
	// 	}
	// 	fprintf(file, "\n");
	// }

	for (i = 0; i < ny; i++){
		for (j = 0; j < nx; j++){
			// 73 pecision ..
			fprintf(file, "%.73lf ", *(p + i*nx + j));
		}
		fprintf(file, "\n");
	}

	fclose(file);
}

void print_array(double *arr){
	printf("\n\n");	
	int i, j;
	for (i = 0; i < ny; i++){
		for (j = 0; j < nx; j++){
			printf("%e ", *(arr + i * nx + j));
		}
		printf("\n\n");
	}
}


void init(double *u, double *v, double *p, double *b){
	int i, j;
	for (i = 0; i < ny; i++){
		for (j = 0; j < nx; j++){
			*(u + i * nx + j) = 0.0;
			*(v + i * nx + j) = 0.0;
			*(p + i * nx + j) = 0.0;
			*(b + i * nx + j) = 0.0;
		}
	}
}



__global__ void build_up_b(double *b, double *u, double *v, double dx, double dy){


    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < ny - 1 && j > 0 && j < nx - 1){
        *(b + i * nx + j) = rho * (1 / dt *
				((*(u + i * nx + j + 1) - *(u + i * nx + j - 1)) / (2*dx)
				+(*(v + (i+1)*nx + j) - *(v + (i-1)*nx +j)) / (2*dy)) -
			(*(u + i * nx + j + 1) - *(u + i*nx + j-1)) * (*(u + i * nx + j + 1) - *(u + i*nx + j-1)) / (2*2*dx*dx) -
			2 * ((*(u + (i+1)*nx + j) - *(u + (i-1)*nx +j)) / (2*dy) *
			(*(v + i * nx + j + 1) - *(v + i * nx + j - 1)) / (2*dx)) - 
			(*(v + (i+1)*nx + j) - *(v + (i-1)*nx +j)) * (*(v + (i+1)*nx + j) - *(v + (i-1)*nx +j)) / (2*2*dy*dy));
    }


}

__global__ void  solve_pressure_poisson(double *p, double *pn, double *b, double dx, double dy){

	__shared__ double tile[BlockSizeY+2][BlockSizeX+2];

	int ty = threadIdx.y;
    int tx = threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;	


   	if (i < ny && j < nx) {
    	tile[ty+1][tx+1] = *(pn +i*nx+j);
    }
    if (ty == 0 && i > 0) {
        tile[ty][tx+1] = *(pn+(i-1)*nx+j); 
    }
    if (ty == BlockSizeY-1 && i < ny-1) {
        tile[ty+2][tx+1] = *(pn + (i+1)*nx +j);
    }
    if (tx == 0 && j > 0) {
        tile[ty+1][tx] = *(pn + i*nx+j-1);
    }
    if (tx == BlockSizeX-1 && j < nx-1) {
        tile[ty+1][tx+2] = *(pn + i*nx+j+1);
    }

    __syncthreads();


    if (i > 0 && i < ny - 1 && j > 0 && j < nx - 1){
        double tmp = ((tile[ty+1][tx+2] +tile[ty+1][tx]) * dy*dy  + 
                        (tile[ty+2][tx+1] + tile[ty][tx+1]) * dx*dx)/
                        (2 * (dx*dx + dy*dy))- 
                        dx*dx*dy*dy / (2 * (dx*dx + dy*dy)) * *(b + i*nx +j);

        tile[ty+1][tx+1] = tmp;
    }
	__syncthreads();


	if (j == nx-1){
        tile[ty+1][tx+1] = tile[ty+1][tx];
    }

    if (i == 0){
    	tile[ty+1][tx+1] = tile[ty+2][tx+1];	
    }

    if (j == 0){
        tile[ty+1][tx+1] = tile[ty+1][tx+2];
    }

    if (i == ny - 1){
        tile[ty+1][tx+1] = 0;
    }

	__syncthreads();

	if (i < ny && j < nx) {
		*(p + i*nx + j) = tile[ty+1][tx+1];
	}

	
}


__global__ void velocity_update(double *u, double *v, double *p, double dx, double dy){

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("%d ", i);

    // if ( i == ny - 1){
    // 	printf("hello");
    // }

    double uij, uijp1, uip1j, uijm1, uim1j, vij, vijp1, vip1j, vijm1, vim1j;

    if (i > 0 && i < ny - 1 && j > 0 && j < nx - 1){


        uij   = *(u + i*nx + j);
        uijp1 = *(u + i*nx + j + 1);
        uip1j = *(u + (i+1)*nx + j);
        uijm1 = *(u + i*nx + j - 1);
        uim1j = *(u + (i-1)*nx + j);

        vij   = *(v + i*nx + j);
        vijp1 = *(v + i*nx + j + 1);
        vip1j = *(v + (i+1)*nx + j);
        vijm1 = *(v + i*nx + j - 1);
        vim1j = *(v + (i-1)*nx + j);


        *(u + i*nx + j) = uij - uij * dt / dx  * 
								(uij - uijm1) -
								vij * dt / dy *
								(uij - uim1j) - 
								dt / (2 * rho * dx) * (*(p + i*nx + j+1) - *(p + i*nx +j-1)) +
								nu * (dt / (dx*dx) *
								(uijp1 - 2 * uij + uijm1) +
								dt / (dy*dy) *
								(uip1j - 2 * uij + uim1j));

        *(v + i*nx + j) = vij - uij * dt / dx  * 
                                (vij - vijm1) -
                           		vij * dt / dy *
                            	(vij -vim1j) - 
                            	dt / (2 * rho * dy) * (*(p + (i+1)*nx + j) - *(p + (i-1)*nx +j)) +
                            	nu * (dt / (dx*dx) *
                            	(vijp1 - 2 * vij + vijm1) +
                            	dt/ (dy*dy) *
                            	(vip1j - 2 * vij + vim1j));


    }

    if (j == 0){

        *(u + i *nx) = 0;
   		*(v + i *nx) = 0;
    }
    
    if (j == nx - 1){
 		*(u + i * nx + nx - 1) = 0;
 		*(v + i * nx + nx - 1) = 0;
      
    }

    if (i == 0){
        *(u + j) = 0;
 		*(v + j) = 0;
    }

    if (i == ny - 1){
     	*(u + (ny-1)*nx + j) = 1; 	// set velocity on cavity lid equal to 1	
		*(v + (ny-1)*nx + j) = 0; 

    }


}


int main(){

	char* result_file_name = (char *)"flow_results_cuda.txt";

	int n, it;

	double *ucpu, *vcpu, *pcpu, *bcpu, *pncpu;
	double dx = xmax / (nx-1);
	double dy = ymax / (ny-1);


	struct timeval time_start;
    struct timeval time_end;

	ucpu = (double *) malloc((nx * ny) * sizeof(double));
	vcpu = (double *) malloc((nx * ny) * sizeof(double));
	pcpu = (double *) malloc((nx * ny) * sizeof(double));
	pncpu = (double *) malloc((nx * ny) * sizeof(double));
	bcpu = (double *) malloc((nx * ny) * sizeof(double));

	init(ucpu, vcpu, pcpu, bcpu);

	gettimeofday(&time_start, NULL);	


  	double *ugpu, *vgpu, *pgpu, *bgpu, *pngpu, *tempgpu;
    
    hipMalloc((void **)&ugpu, (nx * ny) * sizeof(double));
    hipMalloc((void **)&vgpu, (nx * ny) * sizeof(double));
    hipMalloc((void **)&pgpu, (nx * ny) * sizeof(double));
    hipMalloc((void **)&pngpu, (nx * ny) * sizeof(double));
    hipMalloc((void **)&tempgpu, (nx * ny) * sizeof(double));
    hipMalloc((void **)&bgpu, (nx * ny) * sizeof(double));

    hipMemcpy(ugpu,ucpu, (nx * ny) * sizeof(double),hipMemcpyHostToDevice); 
    hipMemcpy(vgpu,vcpu, (nx * ny) * sizeof(double),hipMemcpyHostToDevice); 
    hipMemcpy(pgpu,pcpu, (nx * ny) * sizeof(double),hipMemcpyHostToDevice); 
    hipMemcpy(pngpu,pncpu, (nx * ny) * sizeof(double),hipMemcpyHostToDevice); 
    hipMemcpy(bgpu,bcpu, (nx * ny) * sizeof(double),hipMemcpyHostToDevice); 

    dim3 dimGrid(GridSizeX, GridSizeY);
    dim3 dimBlock(BlockSizeX, BlockSizeY);


    for(n =0; n < nt; n++){
        build_up_b<<<dimGrid, dimBlock>>>(bgpu, ugpu, vgpu, dx, dy);

	    for (it = 0; it < nit; it++){
	    	tempgpu = pngpu;
	        pngpu = pgpu;
	        pgpu = tempgpu;
	        solve_pressure_poisson<<<dimGrid, dimBlock>>>(pgpu, pngpu, bgpu, dx, dy);

 	  	 }

       velocity_update<<<dimGrid, dimBlock>>>(ugpu, vgpu, pgpu, dx, dy);
	}

    hipMemcpy(ucpu,ugpu, (nx * ny) * sizeof(double),hipMemcpyDeviceToHost); 
    hipMemcpy(vcpu,vgpu, (nx * ny) * sizeof(double),hipMemcpyDeviceToHost); 
    hipMemcpy(pcpu,pgpu, (nx * ny) * sizeof(double),hipMemcpyDeviceToHost); 
    hipMemcpy(bcpu,bgpu, (nx * ny) * sizeof(double),hipMemcpyDeviceToHost); 

    hipFree(ugpu);
    hipFree(vgpu);
    hipFree(pgpu);
    hipFree(bgpu);


   	gettimeofday(&time_end, NULL);

	save_results(ucpu, vcpu, pcpu, result_file_name, dx, dy);

	free(ucpu);
	free(vcpu);
	free(pcpu);
	free(bcpu);


	double exec_time = (double) (time_end.tv_sec - time_start.tv_sec) +
                   (double) (time_end.tv_usec - time_start.tv_usec) / 1000000.0;

    printf("Running time for CUDA code: %lf\n", exec_time);


	return 0;
}